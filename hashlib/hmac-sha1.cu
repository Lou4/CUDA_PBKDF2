#include <stdint.h>
#define IPAD 0x36363636
#define OPAD 0x5c5c5c5c
#include "sha1.cuh"

__device__ void memxor (void * dest, const void * src,size_t n) {

  int rest = n%4;
  n = n/4;
  const int * s = (int*)src;
  int *d = (int*)dest;
  const char * s2 = (char*)src+4*n;
  char *d2 = (char*)dest+4*n;
  for (; n > 0; n--)
    *d++ ^= *s++;
  for (; rest > 0; rest--)
	  *d2++ ^= *s2++;
}
__device__ void hmac_sha1 (const void * key, uint32_t keylen, const void *in, uint32_t inlen, void *resbuf, struct globalChars *chars) {
  struct sha1_ctx inner;
  struct sha1_ctx outer;

  sha1_init_ctx (&inner);
  cudaMemsetDevice (chars->block, IPAD, sizeof (chars->block));
  memxor(chars->block, key, keylen);
  sha1_process_block (chars->block, 64, &inner);
  sha1_process_bytes (in, inlen, &inner);
  sha1_finish_ctx (&inner, chars->innerhash);
  
  /* Compute result from KEY and INNERHASH.  */
  sha1_init_ctx (&outer);
  cudaMemsetDevice (chars->block, OPAD, sizeof (chars->block));
  memxor(chars->block, key, keylen);
  sha1_process_block (chars->block, 64, &outer);
  sha1_process_bytes (chars->innerhash, 20, &outer);   
  sha1_finish_ctx (&outer, resbuf);
}
