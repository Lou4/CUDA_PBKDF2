#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include "my_C_lib/utils.h"
#include "my_C_lib/CPU_time.h"
<<<<<<< HEAD
#include "hashlib/hmac-sha1.cuh"
#include "hashlib/sha1.cuh"

#define H_LEN 20 // Length in Bytes of the PRF functions' output
=======

#define H_LEN 4 // Length in Bytes of the PRF functions' output
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712
#define DEV 0
#define intDivCeil(n, d) ((n + d - 1) / d)

int DEBUG;
char salt[] = "salt";

__constant__ int D_C;
<<<<<<< HEAD
__constant__ int D_SK_LEN;
__constant__ int D_N;


__device__ void actualFunction(char* sk, char* output, int const KERNEL_ID){
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

=======
__constant__ int D_DK_LEN;
__constant__ int D_N;
__device__ char d_salt[] = "salt";	// Static Global Memory

__device__ void actualFunction(char* sk, char* output, int const KERNEL_ID){

	int idx = blockDim.x * blockIdx.x + threadIdx.x;
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712

	if(idx >= D_N)
		return;

<<<<<<< HEAD
	globalChars globalChars;
	uint8_t salt[H_LEN] = "salt";
	int saltLen = 4 + (2 * sizeof(int));
	int *ptr = (int*)&salt[4];
	uint8_t acc[H_LEN];
	uint8_t buffer[H_LEN];


	cudaMemcpyDevice(&ptr[0], &idx, sizeof(int));
	cudaMemcpyDevice(&ptr[1], &KERNEL_ID, sizeof(int));

	/* DEBUG
	if(idx == 1){
		for(int i = 0; i <H_LEN; i++){
			printf("%02x ", salt[i]);
		}
		printf("\n --- \n");
	}*/


	hmac_sha1(sk, D_SK_LEN, salt, saltLen, buffer, &globalChars);
	cudaMemcpyDevice(salt, buffer, H_LEN);
	cudaMemcpyDevice(acc, buffer, H_LEN);
	for(int i = 0; i < D_C; i++){
		hmac_sha1(sk, D_SK_LEN, salt, H_LEN, buffer, &globalChars);
		cudaMemcpyDevice(salt, buffer, H_LEN);

		for(int i = 0; i < H_LEN; i++){
			acc[i] ^= buffer[i];
		}
	}

	/* DEBUG
	if(idx == 0 && KERNEL_ID == 4){
		for(int i = 0; i <H_LEN; i++){
			printf("%02x  ", acc[i]);
		}
		printf("\n --- \n");
	}
	*/
=======
	//uint8_t tmp[D_C][H_LEN];
	char tmp[H_LEN];
	for(int i = 0; i < H_LEN; i++)
		tmp[i] = i*(idx+1);				//TODO: messo per avere tmp non completamente nullo. AL POSTO DI i METTERE 0. Lo scopo di questo ciclo è simulare una memset dove si pulisce l'array


	/*
	 * salt = salt || KERNEL_ID || idx
	 */
	for(int i = 0; i < D_C; i++){
		/* TODO: chiamata alla hash function
		 * 	salt = hash(sk, salt);
		 * 	tmp[i] = salt;
		 */
		printf("");
	}

	/*
	 * uint8_t result[H_LEN];
	 * result = tmp[0];
	 */
	for(int i = 1; i < D_C; i++){
		/**
		 * result ^= tmp[i]
		 */
		printf("");
	}

>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712

	int index;
	for(int i = 0; i < H_LEN; i++){
		index = idx * H_LEN + i;
<<<<<<< HEAD
		output[index] = acc[i];
	}

=======
		output[index] = tmp[i];
	}
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712
}

__global__ void pbkdf2(char* sk, char* output, int *kernelId){
	actualFunction(sk, output, *kernelId);
}


__global__ void pbkdf2_2(char* sk, char* output, int *kernelId){
	actualFunction(sk, output, *kernelId);
}

__global__ void pbkdf2_3(char* sk, char* output){
	actualFunction(sk, output, 0);
}

__global__ void pbkdf2_4(char* sk, char* output, int *kernelId){
	actualFunction(sk, output, *kernelId);
}

<<<<<<< HEAD
__host__ void execution1(const char* SOURCE_KEY, int const C, int const DK_LEN, int const DK_NUM, int const GX, int const BX, int const THREAD_X_KERNEL, struct Data *out);
__host__ void execution2(const char* SOURCE_KEY, int const C, int const DK_LEN, int const DK_NUM, int const GX, int const BX, int const THREAD_X_KERNEL, struct Data *out);
__host__ void execution3(const char* SOURCE_KEY, int const C, int const DK_LEN, int const DK_NUM, int const GX, int const BX, int const THREAD_X_KERNEL, struct Data *out);
__host__ void execution4(const char* SOURCE_KEY, int const C, int const DK_LEN, int const DK_NUM, int const GX, int const BX, int const THREAD_X_KERNEL, struct Data *out, int const nStream, int const INDEX);
__host__ void executionSequential(const char* SOURCE_KEY, int const C, int const DK_LEN, int const DK_NUM, struct Data *out);
__host__ void copyValueFromGlobalMemoryToCPUMemory(uint8_t *keys, uint8_t *output, int const NUM, int const LEN, int const OFFSET);
__host__ void printAllKeys(uint8_t *keys, int const LEN, int const NUM);
__host__ void printHeader(int const DK_NUM, int const DK_LEN, int const  BX);
__host__ void printKernelDebugInfo(int const K_ID, int const THREAD_X_K, int const K_BYTES_GENERATED, int const DK_LEN);

=======
__host__ void execution1(const char* SOURCE_KEY, int const C, int const DK_LEN, int const DK_NUM, int const GX, int const BX, struct Data *out);
__host__ void execution2(const char* SOURCE_KEY, int const C, int const DK_LEN, int const DK_NUM, int const GX, int const BX, struct Data *out);
__host__ void execution3(const char* SOURCE_KEY, int const C, int const DK_LEN, int const DK_NUM, int const GX, int const BX, struct Data *out);
__host__ void execution4(const char* SOURCE_KEY, int const C, int const DK_LEN, int const DK_NUM, int const GX, int const BX, struct Data *out, int const nStream, int const INDEX);
__host__ void executionSequential(const char* SOURCE_KEY, int const C, int const DK_LEN, int const DK_NUM, struct Data *out);
__host__ void copyValueFromGlobalMemoryToCPUMemory(uint8_t *keys, uint8_t *output, int const NUM, int const LEN);
__host__ void printAllKeys(uint8_t *keys, int const LEN, int const NUM);
__host__ void printHeader(int const DK_NUM, int const DK_LEN, int const  BX);
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712
/**
 * DF = PBKDF2(PRF,Password, Salt, c, dkLen)
 *
 * DF = T1 || T2 || ... || Tdklen/hlen
 *
 * Ti = U1 xor U2 xor … xor Uc
 *
 * U1 = PRF(Password, Salt || i);
 * U2 = PRF(Password, U1);
 * U3 = PRF(Password, U2);
 * . . .
 * Uc = PRF(Password, Uc-1);
 *
 * One thread will calculata one Ti.
 */

struct Data{
	uint8_t *keys;
	double elapsedKernel;
	double elapsedGlobal;
};


int main(int c, char **v){
	system("clear");
<<<<<<< HEAD
	printf("\t\t\t\t----------- Authors -------------\n");
	printf("\t\t\t\t| Luca Tagliabue, Marco Predari |\n");
	printf("\t\t\t\t---------------------------------\n\n");
=======
	printf("Authors:\nLuca Tagliabue, Marco Predari\n\n");
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712

	if(c != 7){
		printf("Error !!\n");
		printf("./Project_GPU <Bx> <source_key> <iterations> <len_derived_keys> <num_derived_keys> <DEBUG>\n");

		exit(EXIT_FAILURE);
	}

	//Host var
	int const BX = atoi(v[1]);				// Thread per block
	char const *SOURCE_KEY = v[2];			// Password
<<<<<<< HEAD
	int const SK_LEN = strlen(SOURCE_KEY);	// Password len
=======
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712
	int const C = atoi(v[3]);				// Number of iteration
	int const DK_LEN = atoi(v[4]);			// Derived Keys' length
	int const DK_NUM = atoi(v[5]);			// Number of derived keys we'll generate
	DEBUG = atoi(v[6]);

	int foo;

	assert(isPowOfTwo(DK_LEN) == 1);
	assert(isPowOfTwo(DK_NUM) == 1);
<<<<<<< HEAD
=======
	assert(isPowOfTwo(H_LEN) == 1);
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712

	// One kernel generate one dk, one thread generate one Ti
	int threadsPerKernel = intDivCeil(DK_LEN, H_LEN);		// Threads needed
	int Gx = intDivCeil(threadsPerKernel, BX);				// Calculate Gx

	//Output var
	struct Data *out1, *out2, *out3, *out4, *outS;
	out1 = (struct Data*)malloc(sizeof(struct Data));
	out2 = (struct Data*)malloc(sizeof(struct Data));
	out3 = (struct Data*)malloc(sizeof(struct Data));

	out1->keys = (uint8_t*)malloc(DK_NUM * DK_LEN * sizeof(uint8_t*));
	out2->keys = (uint8_t*)malloc(DK_NUM * DK_LEN * sizeof(uint8_t*));
	out3->keys = (uint8_t*)malloc(DK_NUM * DK_LEN * sizeof(uint8_t*));

	int const N_STREAM[] = {2, 4, 8, 16};
	int const S_LEN = 4;
	out4 = (struct Data*)malloc(sizeof(struct Data) * S_LEN);
	for(int i = 0; i < S_LEN; i++){
		out4[i].keys = (uint8_t*)malloc(DK_NUM * DK_LEN * sizeof(uint8_t*));
	}

	outS = (struct Data*)malloc(sizeof(struct Data));
	outS->keys = (uint8_t*)malloc(DK_NUM * DK_LEN * sizeof(uint8_t*));

	if(DEBUG){
		printf("SOURCE_KEY: %s\n", SOURCE_KEY);
		printf("C: %d\n", C);
		printf("DK_LEN: %d\n", DK_LEN);
		printf("DK_NUM: %d\n", DK_NUM);
		printf("H_LEN: %d\n", H_LEN);
	}

	printHeader(DK_NUM, DK_LEN, BX);

	CHECK(hipSetDevice(DEV));

<<<<<<< HEAD
	//Tranfer to CONSTANT MEMORY
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_N), &threadsPerKernel, sizeof(int)));	// Thread per kernel
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_SK_LEN), &SK_LEN, sizeof(int)));		// Source key len
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_C), &C, sizeof(int)));					// Iteration
=======
	hipMemcpyToSymbol(HIP_SYMBOL(D_N), &threadsPerKernel, sizeof(int));
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712

	// Without Stream
	printf("- - - - - - Execution one, more kernel no stream - - - - - -\n");
	printf("\nTotal number of threads required per kernel: %d\n\n", threadsPerKernel);
	double start = seconds();
<<<<<<< HEAD
	execution1(SOURCE_KEY, C, DK_LEN, DK_NUM, Gx, BX, threadsPerKernel, out1);
=======
	execution1(SOURCE_KEY, C, DK_LEN, DK_NUM, Gx, BX, out1);
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712
	out1->elapsedGlobal = seconds() - start;
	printf("- - - - - - - End execution one - - - - - - - - - - - - - -\n");

	printf("\n\n\n* * * **************************************************************************************** * * *\n\n\n");

	printf("Press enter to continue . . .");
	//scanf("%d", &foo);

	printHeader(DK_NUM, DK_LEN, BX);

<<<<<<< HEAD
	//Tranfer to CONSTANT MEMORY
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_N), &threadsPerKernel, sizeof(int)));	// Thread per kernel
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_SK_LEN), &SK_LEN, sizeof(int)));		// Source key len
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_C), &C, sizeof(int)));					// Iteration
=======
	hipMemcpyToSymbol(HIP_SYMBOL(D_N), &threadsPerKernel, sizeof(int));
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712

	// With Stream
	printf("- - - - - - Execution two, with stream - - - - - -\n");
	printf("\nTotal number of threads required per kernel: %d\n\n", threadsPerKernel);
	start = seconds();
<<<<<<< HEAD
	execution2(SOURCE_KEY, C, DK_LEN, DK_NUM, Gx, BX, threadsPerKernel, out2);
=======
	execution2(SOURCE_KEY, C, DK_LEN, DK_NUM, Gx, BX, out2);
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712
	out2->elapsedGlobal = seconds() - start;
	printf("- - - - - - - - End execution two - - - - - - - - \n");


	printf("\n\n\n* * * **************************************************************************************** * * *\n\n\n");

	printf("Press enter to continue . . .");
	//scanf("%d", &foo);

	printHeader(DK_NUM, DK_LEN, BX);

	// One kernel generate ALL dk, one thread generate one Ti
	threadsPerKernel = intDivCeil((DK_LEN * DK_NUM), H_LEN);	// Threads needed
	Gx = intDivCeil(threadsPerKernel, BX);				// Calculate Gx

<<<<<<< HEAD
	//Tranfer to CONSTANT MEMORY
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_N), &threadsPerKernel, sizeof(int)));	// Thread per kernel
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_SK_LEN), &SK_LEN, sizeof(int)));		// Source key len
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_C), &C, sizeof(int)));					// Iteration
=======
	hipMemcpyToSymbol(HIP_SYMBOL(D_N), &threadsPerKernel, sizeof(int));
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712

	printf("- - - - - - Execution three, one kernel no stream - - - - - -\n");
	printf("\nTotal number of threads required per kernel: %d\n\n", threadsPerKernel);
	start = seconds();
<<<<<<< HEAD
	execution3(SOURCE_KEY, C, DK_LEN, DK_NUM, Gx, BX, threadsPerKernel, out3);
=======
	execution3(SOURCE_KEY, C, DK_LEN, DK_NUM, Gx, BX, out3);
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712
	out3->elapsedGlobal = seconds() - start;
	printf("- - - - - - - End execution three - - - - - - - - - - - - - -\n");


	printf("\n\n\n* * * **************************************************************************************** * * *\n\n\n");
	printf("Press enter to continue . . .");
	//scanf("%d", &foo);

	printHeader(DK_NUM, DK_LEN, BX);

	printf("- - - - - - Execution four, rational use of stream - - - - - -\n");
	for(int i = 0; i < S_LEN; i++){
		threadsPerKernel = intDivCeil((DK_LEN * DK_NUM), (H_LEN*N_STREAM[i]));	// Threads needed
<<<<<<< HEAD
		Gx = intDivCeil(threadsPerKernel, BX);									// Calculate Gx
		printf("Total Bytes: %d\n", DK_LEN * DK_NUM);
		printf("N_STREAM: %d\n", N_STREAM[i]);
		printf("\nTotal number of threads required per kernel: %d\n", threadsPerKernel);
		printf("Every stream generate %d Bytes.\n\n", threadsPerKernel * H_LEN);

		//Tranfer to CONSTANT MEMORY
		CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_N), &threadsPerKernel, sizeof(int)));	// Thread per kernel
		CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_SK_LEN), &SK_LEN, sizeof(int)));		// Source key len
		CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_C), &C, sizeof(int)));					// Iteration

		start = seconds();
		execution4(SOURCE_KEY, C, DK_LEN, DK_NUM, Gx, BX, threadsPerKernel, out4, N_STREAM[i], i);
=======
		Gx = intDivCeil(threadsPerKernel, BX);				// Calculate Gx
		hipMemcpyToSymbol(HIP_SYMBOL(D_N), &threadsPerKernel, sizeof(int));
		start = seconds();
		execution4(SOURCE_KEY, C, DK_LEN, DK_NUM, Gx, BX, out4, N_STREAM[i], i);
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712
		out4[i].elapsedGlobal = seconds() - start;
		printf("\n\n\n\n");
	}
	printf("- - - - - - - End execution four - - - - - - - - - - - - - -\n");

	printf("\n\n\n* * * **************************************************************************************** * * *\n\n\n");
	printf("Press enter to continue . . .");
	//scanf("%d", &foo);

	printHeader(DK_NUM, DK_LEN, BX);

	printf("- - - - - - Last but not least execution, SEQUENTIAL - - - - - -\n");
	start = seconds();
	executionSequential(SOURCE_KEY, C, DK_LEN, DK_NUM, outS);
	outS->elapsedGlobal = seconds() - start;
	printf("- - - - - - - - End last execution - - - - - - - - - - - - - - -\n");

	printf("\n\n\n* * * **************************************************************************************** * * *\n\n\n");
	printf("Press enter to continue . . .");
	//scanf("%d", &foo);


	// Check correctness first two execution
	printf("check correctness . . .\n");
	for(int i=0; i<DK_NUM; i++){
		if (DEBUG) printf("check %d key (len %d Bytes)...", i, sizeof(uint8_t)*DK_LEN);
		assert(memcmp(out1->keys, out2->keys, DK_NUM * DK_LEN * sizeof(uint8_t)) == 0);
		if (DEBUG) printf("ok\n");
	}

	printf("\n\n\n- - - - - - - - - - RESULT - - - - - - - - - - - - - - \n");
	printf("  Witout stream takes %f millisec\n", out1->elapsedGlobal);
	printf("  With stream takes %f millisec\n", out2->elapsedGlobal);
	printf("  One kernel takes %f millisec\n", out3->elapsedGlobal);
	for(int i = 0; i < S_LEN; i++){
		printf("  Stream with rational takes %f millisec\n", out4[i].elapsedGlobal);
	}
	printf("  Sequential takes %f millisec\n", outS->elapsedGlobal);
	printf("\n");

	printf("  Witout stream kernel and transfert  takes %f millisec\n", out1->elapsedKernel);
	printf("  With stream kernel and transfert takes %f millisec\n", out2->elapsedKernel);
	printf("  One kernel and transfert takes %f millisec\n", out3->elapsedKernel);
	for(int i = 0; i < S_LEN; i++){
		printf("  Stream with kernel and transfert takes %f millisec\n", out4[i].elapsedKernel);
	}
	printf("  Sequential haven't kernel so ... \n");

	printf("\n");

	printf("  With stream gain: %c %2lf\n", 37, 100-((100*out2->elapsedGlobal)/out1->elapsedGlobal));
	printf("  With stream gain kernel and transfert: %c %2lf\n", 37, 100-((100*out2->elapsedKernel)/out1->elapsedKernel));
	printf("  One kernel gain: %c %2lf\n", 37, 100-((100*out3->elapsedGlobal)/out2->elapsedGlobal));
	printf("  One kernel gain kernel and transfert: %c %2lf\n", 37, 100-((100*out3->elapsedKernel)/out2->elapsedKernel));
	printf("- - - - - - - - - - - - - - - - - - - - - - - - - - - -\n");
	return 0;
}



<<<<<<< HEAD
__host__ void execution1(const char* SOURCE_KEY, int const C, int const DK_LEN, int const DK_NUM, int const GX, int const BX, int const THREAD_X_KERNEL, struct Data *out){

	//Alloc and init CPU memory
	int const N_BYTES_OUTPUT =  THREAD_X_KERNEL * H_LEN * DK_NUM * sizeof(char);
=======
__host__ void execution1(const char* SOURCE_KEY, int const C, int const DK_LEN, int const DK_NUM, int const GX, int const BX, struct Data *out){

	//Alloc and init CPU memory
	int const N_BYTES_OUTPUT = DK_LEN * DK_NUM * sizeof(char);
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712
	char	 *output = (char*)malloc(N_BYTES_OUTPUT);
	memset(output, 0, N_BYTES_OUTPUT);

	printf("N_BYTES_OUTPUT: %s Bytes\n", prettyPrintNumber(N_BYTES_OUTPUT));

<<<<<<< HEAD
	checkArchitecturalBoundaries(DEV, GX, 1, BX, 1, N_BYTES_OUTPUT, 0, 0);
=======
	checkArchitecturalBoundaries(DEV, GX, 1, BX, 1, N_BYTES_OUTPUT, 0, 1);
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712

	//Device var
	char *d_output;
	char *d_sk;
	int *d_kernelId;


	dim3 grid(GX, 1, 1);
	dim3 block(BX, 1, 1);


	//- - - ALLOC AND TRANFER TO GLOBAL MEMORY
	CHECK(hipMalloc((void**)&d_kernelId, sizeof(int)));

	// Output var
	CHECK(hipMalloc((void**)&d_output, N_BYTES_OUTPUT));
	CHECK(hipMemset(d_output, 0, N_BYTES_OUTPUT));

	// Source Key
	int N_BYTES_SK = (strlen(SOURCE_KEY) + 1) * sizeof(char); // +1 because of null end char
	CHECK(hipMalloc((void**)&d_sk, N_BYTES_SK));
	CHECK(hipMemcpy(d_sk, SOURCE_KEY, N_BYTES_SK, hipMemcpyHostToDevice));

<<<<<<< HEAD
=======
	//- - - TRANFER ON CONSTANT MEMORY
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_C), &C, sizeof(int)));				// Iteration
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_DK_LEN), &DK_LEN, sizeof(int)));	// Desired Keys Length
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712

	printf("grid(%d, %d, %d) - block(%d, %d, %d)\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);

	//Starting kernel
<<<<<<< HEAD
	printf("Starting %d kernels with %s threads each (%d threads needed)...\n", DK_NUM, prettyPrintNumber(block.x*grid.x), THREAD_X_KERNEL);
	int index;
	double start = seconds();
	for(int i = 0; i < DK_NUM; i++){
		index = i* THREAD_X_KERNEL * H_LEN;
		printKernelDebugInfo(i, THREAD_X_KERNEL, THREAD_X_KERNEL*H_LEN, DK_LEN);

		CHECK(hipMemcpy(d_kernelId, &i, sizeof(int), hipMemcpyHostToDevice));
		pbkdf2<<<grid, block>>>(d_sk, &d_output[index], d_kernelId);
		CHECK(hipMemcpy(&output[index], &d_output[index], DK_LEN * sizeof(char), hipMemcpyDeviceToHost));

		printf("Copy %d° key, %d Bytes starting index output[%d]\n\n", i+1, DK_LEN*sizeof(char), index);
=======
	printf("Starting %d kernels with %s threads each ...\n", DK_NUM, prettyPrintNumber(block.x*grid.x));
	int index;
	double start = seconds();
	for(int i = 0; i < DK_NUM; i++){
		index = i*DK_LEN;
		CHECK(hipMemcpy(d_kernelId, &i, sizeof(int), hipMemcpyHostToDevice));
		pbkdf2<<<grid, block>>>(d_sk, &d_output[index], d_kernelId);
		CHECK(hipMemcpy(&output[index], &d_output[index], DK_LEN * sizeof(char), hipMemcpyDeviceToHost));
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712
	}

	CHECK(hipDeviceSynchronize());

	out->elapsedKernel = seconds() - start;
	printf("%d kernels synchronized ...\n", DK_NUM);

	CHECK(hipDeviceReset());


	// Copy value from output to keys var
<<<<<<< HEAD
	copyValueFromGlobalMemoryToCPUMemory(out->keys, (uint8_t*)output, DK_NUM, DK_LEN, THREAD_X_KERNEL * H_LEN);
=======
	copyValueFromGlobalMemoryToCPUMemory(out->keys, (uint8_t*)output, DK_NUM, DK_LEN);
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712


	// Debug print
	if(DEBUG) printAllKeys(out->keys, DK_LEN, DK_NUM);


}


<<<<<<< HEAD
__host__ void execution2(const char* SOURCE_KEY, int const C, int const DK_LEN, int const DK_NUM, int const GX, int const BX, int const THREAD_X_KERNEL, struct Data *out){
=======
__host__ void execution2(const char* SOURCE_KEY, int const C, int const DK_LEN, int const DK_NUM, int const GX, int const BX, struct Data *out){
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712

	hipDeviceProp_t hipDeviceProp_t;
	hipGetDeviceProperties(&hipDeviceProp_t, DEV);

	printf("hipDeviceProp_t.deviceOverlap: %d\n", hipDeviceProp_t.deviceOverlap);
	assert(hipDeviceProp_t.deviceOverlap != 0);

	//Alloc and init CPU memory
<<<<<<< HEAD
	int const N_BYTES_OUTPUT =  THREAD_X_KERNEL * H_LEN * DK_NUM * sizeof(char);
=======
	int const N_BYTES_OUTPUT = DK_LEN * DK_NUM * sizeof(char);
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712

	char	 *output;
	CHECK(hipHostMalloc((void**)&output, N_BYTES_OUTPUT));
	memset(output, 0, N_BYTES_OUTPUT);

	printf("N_BYTES_OUTPUT: %s Bytes\n", prettyPrintNumber(N_BYTES_OUTPUT));

<<<<<<< HEAD
	checkArchitecturalBoundaries(DEV, GX, 1, BX, 1, N_BYTES_OUTPUT, 0, 0);
=======
	checkArchitecturalBoundaries(DEV, GX, 1, BX, 1, N_BYTES_OUTPUT, 0, 1);
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712


	//Device var
	char *d_output;
	char *d_sk;
	int *d_kernelId;

	dim3 grid(GX, 1, 1);
	dim3 block(BX, 1, 1);


	//- - - ALLOC AND TRANFER TO GLOBAL MEMORY
	hipMalloc((void**)&d_kernelId, DK_NUM*sizeof(int));

	// Output var
	CHECK(hipMalloc((void**)&d_output, N_BYTES_OUTPUT));
	CHECK(hipMemset(d_output, 0, N_BYTES_OUTPUT));

	// Source Key
	int N_BYTES_SK = (strlen(SOURCE_KEY) + 1) * sizeof(char); // +1 because of null end char
	CHECK(hipMalloc((void**)&d_sk, N_BYTES_SK));
	CHECK(hipMemcpy(d_sk, SOURCE_KEY, N_BYTES_SK, hipMemcpyHostToDevice));

<<<<<<< HEAD
=======

	//- - - TRANFER ON CONSTANT MEMORY
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_C), &C, sizeof(int)));				// Iteration
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_DK_LEN), &DK_LEN, sizeof(int)));	// Desired Keys Length

>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712
	printf("grid(%d, %d, %d) - block(%d, %d, %d)\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);


	hipStream_t stream[DK_NUM];

	for(int i = 0; i<DK_NUM; i++){
		CHECK(hipStreamCreate(&stream[i]));
	}

	//Starting kernel
<<<<<<< HEAD
	printf("Starting %d kernels with stream with %s threads each (%d threads needed)...\n", DK_NUM, prettyPrintNumber(block.x*grid.x), THREAD_X_KERNEL);
	int index;
	double start = seconds();
	for(int i = 0; i < DK_NUM; i++){
		index = i * THREAD_X_KERNEL * H_LEN;
		printKernelDebugInfo(i, THREAD_X_KERNEL, THREAD_X_KERNEL*H_LEN, DK_LEN);

		CHECK(hipMemcpyAsync(&d_kernelId[i], &i, sizeof(int), hipMemcpyHostToDevice, stream[i]));
		pbkdf2_2<<<grid, block, 0, stream[i]>>>(d_sk, &d_output[index], &d_kernelId[i]);
		CHECK(hipMemcpyAsync(&output[index], &d_output[index], DK_LEN * sizeof(char), hipMemcpyDeviceToHost, stream[i]));

		printf("Copy %d° key, %d Bytes starting index output[%d]\n\n", i+1, DK_LEN*sizeof(char), index);
=======
	printf("Starting %d kernels with stream with %s threads each...\n", DK_NUM, prettyPrintNumber(block.x*grid.x));
	int index;
	double start = seconds();
	for(int i = 0; i < DK_NUM; i++){
		index = i*DK_LEN;
		CHECK(hipMemcpyAsync(&d_kernelId[i], &i, sizeof(int), hipMemcpyHostToDevice, stream[i]));
		pbkdf2_2<<<grid, block, 0, stream[i]>>>(d_sk, &d_output[index], &d_kernelId[i]);
		CHECK(hipMemcpyAsync(&output[index], &d_output[index], DK_LEN * sizeof(char), hipMemcpyDeviceToHost, stream[i]));
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712
	}

	for(int i = 0; i<DK_NUM; i++){
		CHECK(hipStreamSynchronize(stream[i]));
	}

	out->elapsedKernel = seconds() - start;
	printf("%d stream synchronized ...\n", DK_NUM);

	for(int i = 0; i<DK_NUM; i++){
		CHECK(hipStreamDestroy(stream[i]));
	}

	printf("%d stream destroyed ...\n", DK_NUM);

	// Copy value from output to keys var
<<<<<<< HEAD
	copyValueFromGlobalMemoryToCPUMemory(out->keys, (uint8_t*)output, DK_NUM, DK_LEN, THREAD_X_KERNEL * H_LEN);
=======
	copyValueFromGlobalMemoryToCPUMemory(out->keys, (uint8_t*)output, DK_NUM, DK_LEN);
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712

	// Debug print
	if(DEBUG) printAllKeys(out->keys, DK_LEN, DK_NUM);

	CHECK(hipDeviceReset());

}

<<<<<<< HEAD
__host__ void execution3(const char* SOURCE_KEY, int const C, int const DK_LEN, int const DK_NUM, int const GX, int const BX, int const THREAD_X_KERNEL, struct Data *out){


	int const N_BYTES_OUTPUT =  THREAD_X_KERNEL * H_LEN * sizeof(char);
=======
__host__ void execution3(const char* SOURCE_KEY, int const C, int const DK_LEN, int const DK_NUM, int const GX, int const BX, struct Data *out){


	int const N_BYTES_OUTPUT = DK_LEN * DK_NUM * sizeof(char);
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712
	char	 *output = (char*)malloc(N_BYTES_OUTPUT);
	memset(output, 0, N_BYTES_OUTPUT);

	printf("N_BYTES_OUTPUT: %s Bytes\n", prettyPrintNumber(N_BYTES_OUTPUT));

<<<<<<< HEAD
	checkArchitecturalBoundaries(DEV, GX, 1, BX, 1, N_BYTES_OUTPUT, 0, 0);
=======
	checkArchitecturalBoundaries(DEV, GX, 1, BX, 1, N_BYTES_OUTPUT, 0, 1);
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712

	//Device var
	char *d_output;
	char *d_sk;


	dim3 grid(GX, 1, 1);
	dim3 block(BX, 1, 1);


	//- - - ALLOC AND TRANFER TO GLOBAL MEMORY

	// Output var
	CHECK(hipMalloc((void**)&d_output, N_BYTES_OUTPUT));
	CHECK(hipMemset(d_output, 0, N_BYTES_OUTPUT));

	// Source Key
	int N_BYTES_SK = (strlen(SOURCE_KEY) + 1) * sizeof(char); // +1 because of null end char
	CHECK(hipMalloc((void**)&d_sk, N_BYTES_SK));
	CHECK(hipMemcpy(d_sk, SOURCE_KEY, N_BYTES_SK, hipMemcpyHostToDevice));

<<<<<<< HEAD
	printf("grid(%d, %d, %d) - block(%d, %d, %d)\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);

	//Starting kernel
	printf("Starting ONE kernels with %s threads (%d threads needed)...\n", prettyPrintNumber(block.x * grid.x), THREAD_X_KERNEL);
	double start = seconds();

	pbkdf2_3<<<grid, block>>>(d_sk, d_output);
	CHECK(hipMemcpy(output, d_output, N_BYTES_OUTPUT, hipMemcpyDeviceToHost));

	printf("Copy the all output of %d Bytes compose of %d blocks of %d Bytes\n\n", N_BYTES_OUTPUT, N_BYTES_OUTPUT/H_LEN, H_LEN);
=======

	//- - - TRANFER ON CONSTANT MEMORY
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_C), &C, sizeof(int)));				// Iteration
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_DK_LEN), &DK_LEN, sizeof(int)));	// Desired Keys Length

	printf("grid(%d, %d, %d) - block(%d, %d, %d)\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);

	//Starting kernel
	printf("Starting ONE kernels with %s threads...\n", prettyPrintNumber(block.x * grid.x));
	double start = seconds();

	pbkdf2_3<<<grid, block>>>(d_sk, d_output);

	CHECK(hipMemcpy(output, d_output, N_BYTES_OUTPUT, hipMemcpyDeviceToHost));
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712

	out->elapsedKernel = seconds() - start;
	printf("Kernel synchronized ...\n", DK_NUM);

	CHECK(hipDeviceReset());


	// Copy value from output to keys var
<<<<<<< HEAD
	copyValueFromGlobalMemoryToCPUMemory(out->keys, (uint8_t*)output, DK_NUM, DK_LEN, DK_LEN);
=======
	copyValueFromGlobalMemoryToCPUMemory(out->keys, (uint8_t*)output, DK_NUM, DK_LEN);
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712

	// Debug print
	if(DEBUG) printAllKeys(out->keys, DK_LEN, DK_NUM);

}

<<<<<<< HEAD
__host__ void execution4(const char* SOURCE_KEY, int const C, int const DK_LEN, int const DK_NUM, int const GX, int const BX, int const THREAD_X_KERNEL, struct Data *out, int const N_STREAM, int const INDEX){


	hipDeviceProp_t hipDeviceProp_t;
	hipGetDeviceProperties(&hipDeviceProp_t, DEV);
	assert(hipDeviceProp_t.deviceOverlap != 0);

	int const N_BYTES_OUTPUT = THREAD_X_KERNEL * H_LEN * N_STREAM * sizeof(char);
=======
__host__ void execution4(const char* SOURCE_KEY, int const C, int const DK_LEN, int const DK_NUM, int const GX, int const BX, struct Data *out, int const N_STREAM, int const INDEX){


	hipDeviceProp_t hipDeviceProp_t;
	hipGetDeviceProperties(&hipDeviceProp_t);
	assert(hipDeviceProp_t.deviceOverlap != 0);

	int const N_BYTES_OUTPUT = DK_LEN * DK_NUM * sizeof(char);
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712
	char	 *output;
	hipHostMalloc((void**)&output, N_BYTES_OUTPUT);
	memset(output, 0, N_BYTES_OUTPUT);

	printf("N_BYTES_OUTPUT: %s Bytes\n", prettyPrintNumber(N_BYTES_OUTPUT));

<<<<<<< HEAD
	checkArchitecturalBoundaries(DEV, GX, 1, BX, 1, N_BYTES_OUTPUT, 0, 0);
=======
	checkArchitecturalBoundaries(DEV, GX, 1, BX, 1, N_BYTES_OUTPUT, 0, 1);
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712

	//Device var
	char *d_output;
	char *d_sk;
	int *d_kernelId;

	dim3 grid(GX, 1, 1);
	dim3 block(BX, 1, 1);


	//- - - ALLOC AND TRANFER TO GLOBAL MEMORY
<<<<<<< HEAD
	hipMalloc((void**)&d_kernelId, N_STREAM*sizeof(int));
=======
	hipMalloc((void**)&d_kernelId, DK_NUM*sizeof(int));
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712

	// Output var
	CHECK(hipMalloc((void**)&d_output, N_BYTES_OUTPUT));
	CHECK(hipMemset(d_output, 0, N_BYTES_OUTPUT));

	// Source Key
	int N_BYTES_SK = (strlen(SOURCE_KEY) + 1) * sizeof(char); // +1 because of null end char
	CHECK(hipMalloc((void**)&d_sk, N_BYTES_SK));
	CHECK(hipMemcpy(d_sk, SOURCE_KEY, N_BYTES_SK, hipMemcpyHostToDevice));

<<<<<<< HEAD
=======

	//- - - TRANFER ON CONSTANT MEMORY
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_C), &C, sizeof(int)));				// Iteration
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_DK_LEN), &DK_LEN, sizeof(int)));	// Desired Keys Length

>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712
	printf("grid(%d, %d, %d) - block(%d, %d, %d)\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);



	hipStream_t stream[N_STREAM];

	for(int i = 0; i < N_STREAM; i++){
		hipStreamCreate(&stream[i]);
	}

	//Starting kernel
<<<<<<< HEAD
	printf("Starting %d kernel with stream with %s threads each (%d threads needed)...\n", N_STREAM, prettyPrintNumber(grid.x * block.x), THREAD_X_KERNEL);
=======
	printf("Starting %d stream with %d threads each...\n", N_STREAM, grid.x * block.x);
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712

	int nBytes;
	int oIndex;
	double start = seconds();
	for(int i = 0; i < N_STREAM; i++){
<<<<<<< HEAD
		nBytes = THREAD_X_KERNEL * H_LEN;
		oIndex = i*nBytes;
		printKernelDebugInfo(i, THREAD_X_KERNEL, nBytes, DK_LEN);

		CHECK(hipMemcpyAsync(&d_kernelId[i], &i, sizeof(int), hipMemcpyHostToDevice, stream[i]));
		pbkdf2_4<<<grid, block>>>(d_sk, &d_output[oIndex], &d_kernelId[i]);
		CHECK(hipMemcpyAsync(&output[oIndex], &d_output[oIndex], nBytes * sizeof(char), hipMemcpyDeviceToHost, stream[i]));

		printf("Copy %d° macro-block of %d Bytes, starting at index output[%d]\n\n", i+1, nBytes, oIndex);
=======
		nBytes = DK_LEN * DK_NUM / N_STREAM;
		oIndex = i*nBytes;
		printf("(%d) %d stream handle %d Bytes of %d\n",oIndex, i, nBytes, N_BYTES_OUTPUT);
		CHECK(hipMemcpyAsync(&d_kernelId[i], &i, sizeof(int), hipMemcpyHostToDevice, stream[i]));
		pbkdf2_4<<<grid, block>>>(d_sk, d_output, d_kernelId);
		CHECK(hipMemcpyAsync(&output[oIndex], &d_output[oIndex], nBytes * sizeof(char), hipMemcpyDeviceToHost, stream[i]));
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712
	}

	for(int i = 0; i < N_STREAM; i++){
		hipStreamSynchronize(stream[i]);
	}

	out[INDEX].elapsedKernel = seconds() - start;
	printf("%d stream synchronized...\n", N_STREAM);


	for(int i = 0; i < N_STREAM; i++){
		hipStreamDestroy(stream[i]);
	}
	printf("%d stream destroyed...\n", N_STREAM);

<<<<<<< HEAD

	// Copy value from output to keys var
	copyValueFromGlobalMemoryToCPUMemory(out[INDEX].keys, (uint8_t*)output, DK_NUM, DK_LEN, DK_LEN);
=======
	CHECK(hipMemcpy(output, d_output, N_BYTES_OUTPUT, hipMemcpyDeviceToHost));

	// Copy value from output to keys var
	copyValueFromGlobalMemoryToCPUMemory(out[INDEX].keys, (uint8_t*)output, DK_NUM, DK_LEN);
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712

	CHECK(hipDeviceReset());

	// Debug print
<<<<<<< HEAD
	if(DEBUG) printAllKeys(out[INDEX].keys, DK_LEN, DK_NUM);
=======
	if(DEBUG) printAllKeys(out->keys, DK_LEN, DK_NUM);
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712
}

__host__ void executionSequential(const char* SOURCE_KEY, int const C, int const DK_LEN, int const DK_NUM, struct Data *out){


	printf("Chiavi: %d\nBlocchi: %d\nIterazioni: %d\n", DK_NUM, DK_LEN/H_LEN, C);

	uint8_t tmp[H_LEN];
	uint8_t output[DK_LEN*DK_NUM];
	for(int numKey = 0; numKey < DK_NUM; numKey++){
<<<<<<< HEAD
		for(int block = 0; block < intDivCeil(DK_LEN , H_LEN); block++){
=======
		for(int block = 0; block < DK_LEN / H_LEN; block++){
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712
			for(int iteration = 0; iteration < C; iteration++){
				/**
				 * Predosky, you're up !!!!
				 */

				//FOO GENERATOR: predosky delete it.
				for(int i = 0; i < H_LEN; i++){
					tmp[i] = i+1;
				}
			}
			memcpy(&output[numKey*H_LEN], tmp, H_LEN * sizeof(uint8_t));
			memset(tmp, 0, H_LEN*sizeof(uint8_t));
		}

	}

	//out->key is a linear matrix
	memcpy(out->keys, output, DK_LEN * DK_NUM * sizeof(uint8_t));
}


<<<<<<< HEAD
__host__ void copyValueFromGlobalMemoryToCPUMemory(uint8_t *keys, uint8_t *output, int const NUM, int const LEN, int const OFFSET){
	for(int i = 0, j = 0; i < NUM; i++, j += OFFSET){
=======
__host__ void copyValueFromGlobalMemoryToCPUMemory(uint8_t *keys, uint8_t *output, int const NUM, int const LEN){
	for(int i = 0, j = 0; i < NUM; i++, j += LEN){
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712
		memcpy(&keys[i*LEN], &output[j], LEN);
	}
}

__host__ void printAllKeys(uint8_t *keys, int const LEN, int const NUM){
	int index;
	for(int i=0; i<NUM; i++){
		printf("(%d° key): ", i);
		for(int j=0; j<LEN; j++){
			index = (i * LEN) + j;
			printf("%02x ", keys[index]);
		}
<<<<<<< HEAD
		printf("\n\n");
=======
		printf("\n");
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712
	}
}

__host__ void printHeader(int const DK_NUM, int const DK_LEN, int const  BX){
	printf("\n- - - - REQUEST - - - - -  \n");
	printf("| %d Keys.\t\t |\n", DK_NUM);
	printf("| %d Bytes per Key.\t |\n", DK_LEN);
	printf("| %d Threads per block.\t |\n", BX);
	printf("| %d Byte H_LEN. \t |\n", H_LEN);
	printf("- - - - - - - - - - - - - \n\n");
}
<<<<<<< HEAD

__host__ void printKernelDebugInfo(int const K_ID, int const THREAD_X_K, int const K_BYTES_GENERATED, int const DK_LEN){
	printf("%d° kernel, %d thread, generate %d Bytes (%d Bytes each block), derived key len %d\n", K_ID+1, THREAD_X_K, K_BYTES_GENERATED, H_LEN, DK_LEN);
}
=======
>>>>>>> bfbfddfc730f8b0abf1d8d07500a0d9da9264712
