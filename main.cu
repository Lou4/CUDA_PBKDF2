#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hiprand/hiprand_kernel.h>
#include "my_C_lib/utils.h"
#include "my_C_lib/CPU_time.h"
#include "hashlib/hmac-sha1.cuh"
#include "hashlib/sha1.cuh"
#include "seqlib/seq_hmac_sha1.cuh"


#define H_LEN 20 // Length in Bytes of the PRF functions' output
#define DEV 0
#define intDivCeil(n, d) ((n + d - 1) / d)
#define SK_MAX_LEN 100

int PRINT_KEY, INFO, SLOW_EXE;

__constant__ char D_SK[SK_MAX_LEN];
__constant__ int D_SK_LEN;
__constant__ long D_DK_LEN;
__constant__ int D_C;
__constant__ int D_N;


__device__ void actualFunction(char* output, int const KERNEL_ID, hiprandState *randomStates){
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if(idx >= D_N)
		return;

	globalChars globalChars;
	uint8_t salt[H_LEN] = "salt\0\0\0\0\0\0\0\0\0\0\0\0\0\0\0";
	hiprandState hiprandState;

	int saltLen = 4 + sizeof(float);
	int *ptr = (int*)&salt[4];
	long seed = idx;

	uint8_t acc[H_LEN];
	uint8_t buffer[H_LEN];

	hiprand_init(seed, KERNEL_ID,  0, &hiprandState);

	//attatching salt
	float rr = hiprand_uniform(&hiprandState);

	/* DEBUG
	printf("(%d, %d): %f\n", idx, KERNEL_ID, rr);
	*/

	cudaMemcpyDevice(ptr, &rr, sizeof(float));
	cudaMemcpyDevice(&ptr[1], &D_DK_LEN, sizeof(int));

	if(idx == 0 && KERNEL_ID == 0){
		// BYTES ARE STORED IN BIG ENDIAND
		/*
		uint8_t *foo = (uint8_t*)&rr;
		uint8_t *bar = (uint8_t*)&D_DK_LEN;
		for(int i = 0; i<sizeof(float); i++){
			printf("%02x ", foo[i]);
		}
		printf("\n");
		for(int i = 0; i<sizeof(long); i++){
			printf("%02x ", bar[i]);
		}
		printf("\n");*/
		printf("Salt: ");
		for(int i = 0; i < H_LEN; i++){
			printf("%02x ", salt[i]);
		}
		printf("\n");
	}

	hmac_sha1(D_SK, D_SK_LEN, salt, saltLen, buffer, &globalChars);
	cudaMemcpyDevice(salt, buffer, H_LEN);
	cudaMemcpyDevice(acc, buffer, H_LEN);
	for(int i = 0; i < D_C; i++){
		hmac_sha1(D_SK, D_SK_LEN, salt, H_LEN, buffer, &globalChars);
		cudaMemcpyDevice(salt, buffer, H_LEN);

		for(int i = 0; i < H_LEN; i++){
			acc[i] ^= buffer[i];
		}
	}

	int index;
	for(int i = 0; i < H_LEN; i++){
		index = idx * H_LEN + i;
		output[index] = acc[i];
	}

}

__global__ void pbkdf2(char* output, int *kernelId, hiprandState *randomStates){
	actualFunction(output, *kernelId, randomStates);
}


__global__ void pbkdf2_2(char* output, int *kernelId, hiprandState *randomStates){
	actualFunction(output, *kernelId, randomStates);
}

__global__ void pbkdf2_3(char* output, hiprandState *randomStates){
	actualFunction(output, 0, randomStates);
}

__global__ void pbkdf2_4(char* output, int *kernelId, hiprandState *randomStates){
	actualFunction(output, *kernelId, randomStates);
}

__host__ void execution1(long const DK_LEN, long const DK_NUM, int const GX, int const BX, int const THREAD_X_KERNEL, struct Data *out);
__host__ void execution2(long const DK_LEN, long const DK_NUM, int const GX, int const BX, int const THREAD_X_KERNEL, struct Data *out);
__host__ void execution3(long const DK_LEN, long const DK_NUM, int const GX, int const BX, int const THREAD_X_KERNEL, struct Data *out);
__host__ void execution4(long const DK_LEN, long const DK_NUM, int const GX, int const BX, int const THREAD_X_KERNEL, struct Data *out, int const nStream, int const INDEX);
__host__ void executionSequential(const char* SOURCE_KEY, int const C, long const DK_LEN, long const DK_NUM, struct Data *out);
__host__ void copyValueFromGlobalMemoryToCPUMemory(uint8_t *keys, uint8_t *output, int const NUM, int const LEN, int const OFFSET);
__host__ void printAllKeys(uint8_t *keys, int const LEN, int const NUM);
__host__ void printHeader(long const DK_NUM, long const DK_LEN, int const  BX, int const C);
__host__ void printKernelDebugInfo(int const K_ID, int const THREAD_X_K, int const K_BYTES_GENERATED, int const DK_LEN);

/**
 * DF = PBKDF2(PRF,Password, Salt, c, dkLen)
 *
 * DF = T1 || T2 || ... || Tdklen/hlen
 *
 * Ti = U1 xor U2 xor … xor Uc
 *
 * U1 = PRF(Password, Salt || i);
 * U2 = PRF(Password, U1);
 * U3 = PRF(Password, U2);
 * . . .
 * Uc = PRF(Password, Uc-1);
 *
 * One thread will calculata one Ti.
 */

struct Data{
	uint8_t *keys;
	double elapsedKernel;
	double elapsedGlobal;
};


int main(int c, char **v){
	system("clear");
	printf("\t\t\t\t----------- Authors -------------\n");
	printf("\t\t\t\t| Luca Tagliabue, Marco Predari |\n");
	printf("\t\t\t\t---------------------------------\n\n");

	if(c != 9){
		printf("Error !!\n");
		printf("./Project_GPU <Bx> <source_key> <iterations> <len_derived_keys> <num_derived_keys> <PRINT_KEY> <INFO> <SLOW_EXECUTION>\n");

		exit(EXIT_FAILURE);
	}

	printf("- - - - - - - - REMINDER - - - - - - - - \n");
	printf("|    sizeof(hiprandState): %d Bytes     |\n", sizeof(hiprandState));
	printf("- - - - - - - - - - - - - - - - - - - - ");

	printf("\n\n\n\n");


	//Host var
	int const BX = atoi(v[1]);				// Thread per block
	char const *SOURCE_KEY = v[2];			// Password
	int const SK_LEN = strlen(SOURCE_KEY);	// Password len
	int const C = atoi(v[3]);				// Number of iteration
	long const DK_LEN = atoi(v[4]);			// Derived Keys' length
	long const DK_NUM = atoi(v[5]);			// Number of derived keys we'll generate
	PRINT_KEY = atoi(v[6]);
	INFO = atoi(v[7]);
	SLOW_EXE = atoi(v[8]);

	int foo;

	assert(isPowOfTwo(BX) == 1);
	assert(isPowOfTwo(DK_LEN) == 1);
	assert(isPowOfTwo(DK_NUM) == 1);
	assert(SK_LEN <= SK_MAX_LEN);

	// One kernel generate one dk, one thread generate one Ti
	int *threadsPerKernel;
	threadsPerKernel = (int*) malloc(sizeof(int));
	*threadsPerKernel = intDivCeil(DK_LEN, H_LEN);		// Threads needed
	int Gx = intDivCeil(*threadsPerKernel, BX);			// Calculate Gx

	//Output var
	int const N_STREAM[] = {2, 4, 8, 16};
	int const S_LEN = 4;
	struct Data *out1, *out2, *out3, *out4, *outS;
	out1 = (struct Data*)malloc(sizeof(struct Data));
	out2 = (struct Data*)malloc(sizeof(struct Data));
	out3 = (struct Data*)malloc(sizeof(struct Data));
	out4 = (struct Data*)malloc(sizeof(struct Data) * S_LEN);
	outS = (struct Data*)malloc(sizeof(struct Data));

	out1->keys = (uint8_t*)malloc(DK_NUM * DK_LEN * sizeof(uint8_t*));
	out2->keys = (uint8_t*)malloc(DK_NUM * DK_LEN * sizeof(uint8_t*));
	out3->keys = (uint8_t*)malloc(DK_NUM * DK_LEN * sizeof(uint8_t*));
	for(int i = 0; i < S_LEN; i++)
		out4[i].keys = (uint8_t*)malloc(DK_NUM * DK_LEN * sizeof(uint8_t*));
	outS->keys = (uint8_t*)malloc(DK_NUM * DK_LEN * sizeof(uint8_t*));

	printHeader(DK_NUM, DK_LEN, BX, C);

	CHECK(hipSetDevice(DEV));

	//Tranfer to CONSTANT MEMORY
	int N_BYTES_SK = (strlen(SOURCE_KEY) + 1) * sizeof(char); // +1 because of null end char
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_SK), 		SOURCE_KEY, 			N_BYTES_SK));	// Source Key
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_SK_LEN), 	&SK_LEN, 			sizeof(int)));	// Source key len
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_DK_LEN), 	&DK_LEN, 			sizeof(long)));	// Derived key len
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_C), 		&C, 					sizeof(int)));	// Iteration
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_N), 		threadsPerKernel, 	sizeof(int)));	// Thread per kernel


	// Without Stream
	printf("- - - - - - Execution one, more kernel no stream - - - - - -\n");
	printf("\nKernel: %ld, Thread per Kernel: %s\n\n", DK_NUM, prettyPrintNumber(*threadsPerKernel));
	double start = seconds();
	if(SLOW_EXE) execution1(DK_LEN, DK_NUM, Gx, BX, *threadsPerKernel, out1);
	out1->elapsedGlobal = seconds() - start;
	printf("- - - - - - - End execution one - - - - - - - - - - - - - -\n");

	printf("\n\n\n\n\n\n\n\n\n* * * **************************************************************************************** * * *\n\n\n\n\n\n\n\n\n");

	printf("Press enter to continue . . .");
	//scanf("%d", &foo);

	printHeader(DK_NUM, DK_LEN, BX, C);

	// With Stream
	printf("- - - - - - Execution two, with stream - - - - - -\n");
	printf("\nStream: %ld, Thread per Stream: %s\n\n", DK_NUM, prettyPrintNumber(*threadsPerKernel));
	start = seconds();
	if(SLOW_EXE) execution2(DK_LEN, DK_NUM, Gx, BX, *threadsPerKernel, out2);
	out2->elapsedGlobal = seconds() - start;
	printf("- - - - - - - - End execution two - - - - - - - - \n");


	printf("\n\n\n\n\n\n\n\n\n* * * **************************************************************************************** * * *\n\n\n\n\n\n\n\n\n");

	printf("Press enter to continue . . .");
	//scanf("%d", &foo);

	printHeader(DK_NUM, DK_LEN, BX, C);

	// One kernel generate ALL dk, one thread generate one Ti
	*threadsPerKernel = intDivCeil((DK_LEN * DK_NUM), H_LEN);		// Threads needed
	Gx = intDivCeil(*threadsPerKernel, BX);						// Calculate Gx

	//Tranfer to CONSTANT MEMORY
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_N), threadsPerKernel, sizeof(int)));	// Thread per kernel

	printf("- - - - - - Execution three, one kernel no stream - - - - - -\n");
	printf("\nKernel: 1, Thread per Kernel: %s\n\n", prettyPrintNumber(*threadsPerKernel));
	start = seconds();
	execution3(DK_LEN, DK_NUM, Gx, BX, *threadsPerKernel, out3);
	out3->elapsedGlobal = seconds() - start;
	printf("- - - - - - - End execution three - - - - - - - - - - - - - -\n");


	printf("\n\n\n\n\n\n\n\n\n* * * **************************************************************************************** * * *\n\n\n\n\n\n\n\n\n");

	printf("Press enter to continue . . .");
	//scanf("%d", &foo);

	printHeader(DK_NUM, DK_LEN, BX, C);

	printf("- - - - - - Execution four, rational use of stream - - - - - -\n");
	for(int i = 0; i < S_LEN; i++){
		*threadsPerKernel = intDivCeil((DK_LEN * DK_NUM), (H_LEN*N_STREAM[i]));	// Threads needed

		//Tranfer to CONSTANT MEMORY
		CHECK(hipMemcpyToSymbol(HIP_SYMBOL(D_N), threadsPerKernel, sizeof(int)));	// Thread per kernel

		Gx = intDivCeil(*threadsPerKernel, BX);									// Calculate Gx
		printf("\nStream: %d, Threads per Stream: %s\n", N_STREAM[i], prettyPrintNumber(*threadsPerKernel));
		printf("Every stream generate %s Bytes.\n\n", prettyPrintNumber(*threadsPerKernel * H_LEN));


		start = seconds();
		if(SLOW_EXE) execution4(DK_LEN, DK_NUM, Gx, BX, *threadsPerKernel, out4, N_STREAM[i], i);
		out4[i].elapsedGlobal = seconds() - start;
		printf("\n\n\n\n");
	}
	printf("- - - - - - - End execution four - - - - - - - - - - - - - -\n");

	printf("\n\n\n\n\n\n\n\n\n* * * **************************************************************************************** * * *\n\n\n\n\n\n\n\n\n");

	printf("Press enter to continue . . .");
	//scanf("%d", &foo);

	printHeader(DK_NUM, DK_LEN, BX, C);

	printf("- - - - - - Last but not least execution, SEQUENTIAL - - - - - -\n");
	start = seconds();
	executionSequential(SOURCE_KEY, C, DK_LEN, DK_NUM, outS);
	outS->elapsedGlobal = seconds() - start;
	printf("- - - - - - - - End last execution - - - - - - - - - - - - - - -\n");

	printf("\n\n\n\n\n\n\n\n\n* * * **************************************************************************************** * * *\n\n\n\n\n\n\n\n\n");

	printf("Press enter to continue . . .");
	//scanf("%d", &foo);


	// Check correctness first two execution
	printf("check correctness . . .\n");
	for(int i=0; i<DK_NUM; i++){
		if (INFO) printf("check %d key (len %d Bytes)...", i, sizeof(uint8_t)*DK_LEN);
		assert(memcmp(out1->keys, out2->keys, DK_NUM * DK_LEN * sizeof(uint8_t)) == 0);
		if (INFO) printf("ok\n");
	}

	printf("\n\n\n- - - - - - - - - - RESULT - - - - - - - - - - - - - - \n");
	printf("  One kernel per key takes \t %f seconds\n", out1->elapsedGlobal);
	printf("  One stream per key takes \t %f seconds\n", out2->elapsedGlobal);
	printf("  One kernel takes \t %f seconds\n", out3->elapsedGlobal);
	for(int i = 0; i < S_LEN; i++){
		printf("  %d Stream takes \t %f seconds\n", N_STREAM[i], out4[i].elapsedGlobal);
	}
	printf("  Sequential takes \t %f seconds\n", outS->elapsedGlobal);
	printf("\n");


	printf("  One kernel vs One kernel per key: \t %c %2lf\n", 37, 100-((100*out3->elapsedGlobal)/out1->elapsedGlobal));
	printf("  One kernel vs One stream per key: \t %c %2lf\n", 37, 100-((100*out3->elapsedGlobal)/out2->elapsedGlobal));
	for(int i = 0; i < S_LEN; i++){
		printf("  One kernel vs %d stream: \t %c %2lf\n", N_STREAM[i], 37, 100-((100*out3->elapsedGlobal)/out4[i].elapsedGlobal));
	}
	printf("  One kernel vs Sequential: \t %c %2lf\n", 37, 100-((100*out3->elapsedGlobal)/outS->elapsedGlobal));
	printf("- - - - - - - - - - - - - - - - - - - - - - - - - - - -\n");

	hipDeviceReset();

	return 0;
}



__host__ void execution1(long const DK_LEN, long const DK_NUM, int const GX, int const BX, int const THREAD_X_KERNEL, struct Data *out){

	//Alloc and init CPU memory
	long const N_BYTES_OUTPUT =  (long)THREAD_X_KERNEL * H_LEN * DK_NUM * sizeof(char);
	long const N_BYTES_CURAND_STATE = (long)THREAD_X_KERNEL * sizeof(hiprandState);

	printf("Global memory required: %s Bytes (%s Bytes for keys + %s Bytes for hiprandState)\n", prettyPrintNumber(N_BYTES_OUTPUT + N_BYTES_CURAND_STATE), prettyPrintNumber(N_BYTES_OUTPUT), prettyPrintNumber(N_BYTES_CURAND_STATE));
	printf("Total length of the keys: %s Bytes (overhead %s Bytes)\n", prettyPrintNumber(DK_LEN * DK_NUM), prettyPrintNumber(N_BYTES_OUTPUT - (DK_LEN * DK_NUM)));
	checkArchitecturalBoundaries(DEV, GX, 1, BX, 1, N_BYTES_OUTPUT + N_BYTES_CURAND_STATE, 0, INFO);

	//Device var
	char *d_output;
	int *d_kernelId;
	hiprandState *d_randomStates;

	dim3 grid(GX, 1, 1);
	dim3 block(BX, 1, 1);

	//- - - ALLOC AND TRANFER TO GLOBAL MEMORY
	CHECK(hipMalloc((void**)&d_kernelId, sizeof(int)));
	CHECK(hipMalloc((void**)&d_output, N_BYTES_OUTPUT));
	CHECK(hipMalloc((void**)&d_randomStates, N_BYTES_CURAND_STATE));
	CHECK(hipMemset(d_output, 0, N_BYTES_OUTPUT));
	CHECK(hipMemset(d_kernelId, 0, sizeof(int)));


	printf("grid(%d, %d, %d) - block(%d, %d, %d)\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);

	//Starting kernel
	long index;
	double start = seconds();
	time_t t;
	struct tm *timestamp;
	for(int i = 0; i < DK_NUM; i++){
		index = (long)i * THREAD_X_KERNEL * H_LEN;
		if(INFO) printKernelDebugInfo(i, THREAD_X_KERNEL, THREAD_X_KERNEL*H_LEN, DK_LEN);

		CHECK(hipMemcpy(d_kernelId, &i, sizeof(int), hipMemcpyHostToDevice));
		pbkdf2<<<grid, block>>>(&d_output[index], d_kernelId, d_randomStates);
		CHECK(hipMemcpy(&out->keys[i*DK_LEN], &d_output[index], DK_LEN * sizeof(char), hipMemcpyDeviceToHost));

		// Completness
		t = time(NULL);
		timestamp = gmtime(&t);
		printf("%d° kernel of %d done . . . [%dh %dmin %dsec UTC]\n", i+1, DK_NUM, timestamp->tm_hour, timestamp->tm_min, timestamp->tm_sec);

		if(INFO) printf("Copy %d° key, %d Bytes starting index output[%ld]\n\n", i+1, DK_LEN*sizeof(char), index);
	}

	CHECK(hipDeviceSynchronize());
	out->elapsedKernel = seconds() - start;

	printf("\n");

	if(INFO) printf("%d kernels synchronized ...\n", DK_NUM);
	if(PRINT_KEY) printAllKeys(out->keys, DK_LEN, DK_NUM);

	CHECK(hipFree(d_output));
	CHECK(hipFree(d_kernelId));
	CHECK(hipFree(d_randomStates));
}

__host__ void execution2(long const DK_LEN, long const DK_NUM, int const GX, int const BX, int const THREAD_X_KERNEL, struct Data *out){

	hipDeviceProp_t hipDeviceProp_t;
	hipGetDeviceProperties(&hipDeviceProp_t, DEV);

	if(INFO) printf("hipDeviceProp_t.deviceOverlap: %d\n", hipDeviceProp_t.deviceOverlap);
	assert(hipDeviceProp_t.deviceOverlap != 0);

	//Alloc and init CPU memory
	char	 *output;
	int *kid;
	long const N_BYTES_OUTPUT =  (long)THREAD_X_KERNEL * H_LEN * DK_NUM * sizeof(char);
	long const N_BYTES_CURAND_STATE = (long)THREAD_X_KERNEL * sizeof(hiprandState);
	int const N_BYTES_KID = DK_NUM * sizeof(int);
	CHECK(hipHostMalloc((void**)&output, N_BYTES_OUTPUT));
	CHECK(hipHostMalloc((void**)&kid, N_BYTES_KID));
	memset(output, 0, N_BYTES_OUTPUT);
	memset(kid, 0, N_BYTES_KID);

	printf("Global memory required: %s Bytes (%s Bytes for keys + %s Bytes for hiprandState + %s Bytes for kernel IDs)\n", prettyPrintNumber(N_BYTES_OUTPUT + N_BYTES_CURAND_STATE + N_BYTES_KID), prettyPrintNumber(N_BYTES_OUTPUT), prettyPrintNumber(N_BYTES_CURAND_STATE), prettyPrintNumber(N_BYTES_KID));
	printf("Total length of the keys: %s Bytes (overhead %s Bytes)\n", prettyPrintNumber(DK_LEN * DK_NUM), prettyPrintNumber(N_BYTES_OUTPUT - (DK_LEN * DK_NUM)));
	checkArchitecturalBoundaries(DEV, GX, 1, BX, 1, N_BYTES_OUTPUT + N_BYTES_CURAND_STATE + N_BYTES_KID, 0, INFO);


	//Device var
	char *d_output;
	int *d_kernelId;
	hiprandState *randomStates;

	dim3 grid(GX, 1, 1);
	dim3 block(BX, 1, 1);


	//- - - ALLOC AND TRANFER TO GLOBAL MEMORY
	CHECK(hipMalloc((void**)&d_kernelId, 	N_BYTES_KID));
	CHECK(hipMalloc((void**)&d_output, 		N_BYTES_OUTPUT));
	CHECK(hipMalloc((void**)&randomStates, 	N_BYTES_CURAND_STATE));
	CHECK(hipMemset(d_kernelId,		0, 		N_BYTES_KID));
	CHECK(hipMemset(d_output, 		0,		N_BYTES_OUTPUT));


	printf("grid(%d, %d, %d) - block(%d, %d, %d)\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);

	hipStream_t stream[DK_NUM];
	hipEvent_t event[DK_NUM];
	for(int i = 0; i<DK_NUM; i++){
		CHECK(hipStreamCreate(&stream[i]));
		CHECK(hipEventCreate(&event[i]));
	}

	//Starting kernel
	long index;
	double start = seconds();
	for(int i = 0; i < DK_NUM; i++){
		index = (long)i * THREAD_X_KERNEL * H_LEN;
		if(INFO) printKernelDebugInfo(i, THREAD_X_KERNEL, THREAD_X_KERNEL*H_LEN, DK_LEN);

		kid[i] = i;

		CHECK(hipMemcpyAsync(&d_kernelId[i], &kid[i], sizeof(int), hipMemcpyHostToDevice, stream[i]));
		pbkdf2_2<<<grid, block, 0, stream[i]>>>(&d_output[index], &d_kernelId[i], randomStates);
		CHECK(hipMemcpyAsync(&output[index], &d_output[index], DK_LEN * sizeof(char), hipMemcpyDeviceToHost, stream[i]));
		CHECK(hipEventRecord(event[i], stream[i]));

		if(INFO) printf("Copy %d° key, %d Bytes starting index output[%ld]\n\n", i+1, DK_LEN*sizeof(char), index);
	}

	// Sync and profiling
	time_t t;
	struct tm *timestamp;
	for(int i = 0; i<DK_NUM; i++){
		CHECK(hipEventSynchronize(event[i]));
		t = time(NULL);
		timestamp = gmtime(&t);
		printf("Stream %d complete . . . [%dh %dmin %dsec UTC]\n", i, timestamp->tm_hour, timestamp->tm_min, timestamp->tm_sec);
	}
	out->elapsedKernel = seconds() - start;

	if(INFO) printf("%d stream synchronized ...\n", DK_NUM);

	for(int i = 0; i<DK_NUM; i++){
		CHECK(hipStreamDestroy(stream[i]));
		CHECK(hipEventDestroy(event[i]));
	}

	if(INFO) printf("%d stream destroyed ...\n", DK_NUM);

	// Copy value from output to keys var
	copyValueFromGlobalMemoryToCPUMemory(out->keys, (uint8_t*)output, DK_NUM, DK_LEN, THREAD_X_KERNEL * H_LEN);

	// Debug print
	if(PRINT_KEY) printAllKeys(out->keys, DK_LEN, DK_NUM);

	CHECK(hipHostFree(output));
	CHECK(hipHostFree(kid));
	CHECK(hipFree(d_output));
	CHECK(hipFree(d_kernelId));
	CHECK(hipFree(randomStates));

}

__host__ void execution3(long const DK_LEN, long const DK_NUM, int const GX, int const BX, int const THREAD_X_KERNEL, struct Data *out){

	long const N_BYTES_OUTPUT =  (long)THREAD_X_KERNEL * H_LEN * sizeof(char);
	long const N_BYTES_CURAND_STATE = (long)THREAD_X_KERNEL * sizeof(hiprandState);

	printf("Global memory required: %s Bytes (%s Bytes for keys + %s Bytes for hiprandState)\n", prettyPrintNumber(N_BYTES_OUTPUT + N_BYTES_CURAND_STATE), prettyPrintNumber(N_BYTES_OUTPUT), prettyPrintNumber(N_BYTES_CURAND_STATE));
	printf("Total length of the keys: %s Bytes (overhead %s Bytes)\n", prettyPrintNumber(DK_LEN * DK_NUM), prettyPrintNumber(N_BYTES_OUTPUT - (DK_LEN * DK_NUM)));
	checkArchitecturalBoundaries(DEV, GX, 1, BX, 1, N_BYTES_OUTPUT, 0, INFO);

	//Device var
	char *d_output;
	hiprandState *randomStates;

	dim3 grid(GX, 1, 1);
	dim3 block(BX, 1, 1);


	//- - - ALLOC AND TRANFER TO GLOBAL MEMORY
	CHECK(hipMalloc((void**)&d_output, N_BYTES_OUTPUT));
	CHECK(hipMemset(d_output, 0, N_BYTES_OUTPUT));
	CHECK(hipMalloc((void**)&randomStates, N_BYTES_CURAND_STATE));


	printf("grid(%d, %d, %d) - block(%d, %d, %d)\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);

	//Starting kernel
	double start = seconds();
	pbkdf2_3<<<grid, block>>>(d_output, randomStates);
	CHECK(hipMemcpy(out->keys, d_output, DK_LEN * DK_NUM, hipMemcpyDeviceToHost));

	if(INFO) printf("Copy the all output of %d Bytes compose of %d blocks of %d Bytes\n\n", N_BYTES_OUTPUT, N_BYTES_OUTPUT/H_LEN, H_LEN);

	out->elapsedKernel = seconds() - start;
	if(INFO) printf("Kernel synchronized ...\n", DK_NUM);

	// Debug print
	if(PRINT_KEY) printAllKeys(out->keys, DK_LEN, DK_NUM);

	CHECK(hipFree(d_output));
	CHECK(hipFree(randomStates));
}

__host__ void execution4(long const DK_LEN, long const DK_NUM, int const GX, int const BX, int const THREAD_X_KERNEL, struct Data *out, int const N_STREAM, int const INDEX){

	hipDeviceProp_t hipDeviceProp_t;
	hipGetDeviceProperties(&hipDeviceProp_t, DEV);
	assert(hipDeviceProp_t.deviceOverlap != 0);

	long const N_BYTES_OUTPUT = (long)THREAD_X_KERNEL * H_LEN * N_STREAM * sizeof(char);
	long const N_BYTES_CURAND_STATE = (long)THREAD_X_KERNEL * sizeof(hiprandState);
	int const N_BYTES_KID = N_STREAM * sizeof(int);
	char	 *output;
	int *kid;
	CHECK(hipHostMalloc((void**)&output, N_BYTES_OUTPUT));
	CHECK(hipHostMalloc((void**)&kid, N_BYTES_KID));
	memset(kid, 0, N_BYTES_KID);
	memset(output, 0, N_BYTES_OUTPUT);

	printf("Global memory required: %s Bytes (%s Bytes for keys + %s Bytes for hiprandState + %s Bytes for kernel IDs)\n", prettyPrintNumber(N_BYTES_OUTPUT + N_BYTES_CURAND_STATE + N_BYTES_KID), prettyPrintNumber(N_BYTES_OUTPUT), prettyPrintNumber(N_BYTES_CURAND_STATE), prettyPrintNumber(N_BYTES_KID));
	printf("Total length of the keys: %s Bytes (overhead %s Bytes)\n", prettyPrintNumber(DK_LEN * DK_NUM), prettyPrintNumber(N_BYTES_OUTPUT - (DK_LEN * DK_NUM)));
	checkArchitecturalBoundaries(DEV, GX, 1, BX, 1, N_BYTES_OUTPUT, 0, INFO);

	//Device var
	char *d_output;
	int *d_kernelId;
	hiprandState *randomStates;

	dim3 grid(GX, 1, 1);
	dim3 block(BX, 1, 1);


	//- - - ALLOC AND TRANFER TO GLOBAL MEMORY
	CHECK(hipMalloc((void**)&d_kernelId, 	N_BYTES_KID));
	CHECK(hipMalloc((void**)&d_output, 		N_BYTES_OUTPUT));
	CHECK(hipMalloc((void**)&randomStates, 	N_BYTES_CURAND_STATE));
	CHECK(hipMemset(d_kernelId, 	0, 	N_BYTES_KID));
	CHECK(hipMemset(d_output, 		0, 	N_BYTES_OUTPUT));



	printf("grid(%d, %d, %d) - block(%d, %d, %d)\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);


	hipStream_t stream[N_STREAM];
	hipEvent_t startEvent[N_STREAM];
	hipEvent_t stopEvent[N_STREAM];
	for(int i = 0; i < N_STREAM; i++){
		CHECK(hipStreamCreate(&stream[i]));
		CHECK(hipEventCreate(&startEvent[i]));
		CHECK(hipEventCreate(&stopEvent[i]));
	}

	//Starting kernel
	long nBytes;
	long oIndex;
	double start = seconds();
	for(int i = 0; i < N_STREAM; i++){
		nBytes = (long)THREAD_X_KERNEL * H_LEN;
		oIndex = nBytes * i;
		if(INFO) printKernelDebugInfo(i, THREAD_X_KERNEL, nBytes, DK_LEN);

		kid[i] = i;

		CHECK(hipEventRecord(startEvent[i]));
		CHECK(hipMemcpyAsync(&d_kernelId[i], &kid[i], sizeof(int), hipMemcpyHostToDevice, stream[i]));
		pbkdf2_4<<<grid, block, 0, stream[i]>>>(&d_output[oIndex], &d_kernelId[i], randomStates);
		CHECK(hipMemcpyAsync(&output[oIndex], &d_output[oIndex], nBytes * sizeof(char), hipMemcpyDeviceToHost, stream[i]));
		CHECK(hipEventRecord(stopEvent[i]));

		if(INFO) printf("Copy %d° macro-block of %d Bytes, starting at index output[%d]\n\n", i+1, nBytes, oIndex);
	}

	// Sync and profiling
	time_t t;
	struct tm *timestamp;
	float elapsed;
	for(int i = 0; i<N_STREAM; i++){
		CHECK(hipEventSynchronize(stopEvent[i]));
		t = time(NULL);
		timestamp = gmtime(&t);
		hipEventElapsedTime(&elapsed, startEvent[i], stopEvent[i]);
		printf("Stream %d complete (elapsed %f millisec). . . [%dh %dmin %dsec UTC]\n", i, elapsed, timestamp->tm_hour, timestamp->tm_min, timestamp->tm_sec);
	}
	out[INDEX].elapsedKernel = seconds() - start;

	if(INFO) printf("%d stream synchronized...\n", N_STREAM);

	for(int i = 0; i < N_STREAM; i++){
		CHECK(hipStreamDestroy(stream[i]));
		CHECK(hipEventDestroy(startEvent[i]));
		CHECK(hipEventDestroy(stopEvent[i]));
	}

	if(INFO) printf("%d stream destroyed...\n", N_STREAM);


	// Copy value from output to keys var
	copyValueFromGlobalMemoryToCPUMemory(out[INDEX].keys, (uint8_t*)output, DK_NUM, DK_LEN, DK_LEN);

	// Debug print
	if(PRINT_KEY) printAllKeys(out[INDEX].keys, DK_LEN, DK_NUM);

	CHECK(hipHostFree(output));
	CHECK(hipHostFree(kid));
	CHECK(hipFree(d_output));
	CHECK(hipFree(d_kernelId));
	CHECK(hipFree(randomStates));
}

__host__ void executionSequential(const char* SOURCE_KEY, int const TOTAL_ITERATIONS, long const DK_LEN, long const DK_NUM, struct Data *out){

	//srand(time(NULL));
	const int NUM_BLOCKS = intDivCeil(DK_LEN, H_LEN);

	printf("Chiavi: %d\nBlocchi: %d\nIterazioni: %d\n", DK_NUM, NUM_BLOCKS, TOTAL_ITERATIONS);

	char salt[H_LEN] = "salt";
	uint8_t buffer[H_LEN];
	uint8_t k_xor[H_LEN];
	int *ptr = (int*) &salt[strlen(salt)];
	const unsigned int sk_len = strlen(SOURCE_KEY);
	const unsigned int salt_len = strlen(salt);

	memset(ptr, 0, H_LEN - strlen(salt));

	if (INFO) {
		printf("Source Key: %s | len : %d\n", SOURCE_KEY, sk_len);
		printf("Total Iterations: %d\n", TOTAL_ITERATIONS);
		printf("Nun Blocks: %d\n", NUM_BLOCKS);
	}

	time_t t;
	struct tm *timestamp;
	int x = 0;
	int total = DK_NUM * NUM_BLOCKS * TOTAL_ITERATIONS;
	int tenPercent = total / 10;

	t = time(NULL);
	timestamp = gmtime(&t);
	if(tenPercent != 0) printf("%c 0 complete . . . [%dh %dmin %dsec UTC]\n", 37, timestamp->tm_hour, timestamp->tm_min, timestamp->tm_sec);
	for(int numKey = 0; numKey < DK_NUM; numKey++) {

		uint8_t acc_key[NUM_BLOCKS * H_LEN];

		for(int block = 0; block < NUM_BLOCKS; block++) {
			//copy the well know salt value
			//memcpy(buffer, salt, salt_len);
			//concatenate values to add entropy to the salt
			ptr[0] = rand();
			memcpy(&ptr[1], &DK_LEN, sizeof(long));

			if (block == 0 && numKey == 0) {
				printf("(key %d) Salt: ", numKey);
				for (int i = 0; i<H_LEN; i++) {
					printf("%02x ", (uint8_t)salt[i]);
				}
				printf("\n");
			}

			//calculate the fist hmac_sha1
			lrad_hmac_sha1((const unsigned char*) SOURCE_KEY, sk_len, (const unsigned char*) salt, H_LEN, buffer);
			//init the xor val
			memcpy(k_xor, buffer, H_LEN);
			//copy back to salt array
			memcpy(salt, buffer, H_LEN);
			//apply iterations to hash fn
			for(int iteration = 0; iteration < TOTAL_ITERATIONS; iteration++) {
				//hash again
				lrad_hmac_sha1((const unsigned char*) SOURCE_KEY, sk_len, (const unsigned char*) salt, H_LEN, buffer);
				//copy back to salt array for the next iteration
				memcpy(salt, buffer, H_LEN);
				//to optimize the algorithm directly xor the sha1 obtained
				for(int k = 0; k < H_LEN; k++) {
					k_xor[k] ^= buffer[k];
				}

				// Completeness
				x++;
				t = time(NULL);
				timestamp = gmtime(&t);
				if(tenPercent != 0 && x % tenPercent  == 0) printf("%c %d complete . . . [%dh %dmin %dsec UTC]\n", 37, 10 * (x / tenPercent), timestamp->tm_hour, timestamp->tm_min, timestamp->tm_sec);
			}
			//concatenate the key part
			memcpy(&acc_key[block * H_LEN], k_xor, H_LEN);
		}
		//save generated key
		memcpy(&out->keys[numKey * DK_LEN], acc_key, DK_LEN);
	}
	printf("x: %d\n",x);

	if(PRINT_KEY) printAllKeys(out->keys, DK_LEN, DK_NUM);

}


__host__ void copyValueFromGlobalMemoryToCPUMemory(uint8_t *keys, uint8_t *output, int const NUM, int const LEN, int const OFFSET){
	for(int i = 0, j = 0; i < NUM; i++, j += OFFSET){
		memcpy(&keys[i*LEN], &output[j], LEN);
	}
}

__host__ void printAllKeys(uint8_t *keys, int const LEN, int const NUM){
	int index;
	for(int i=0; i<NUM; i++){
		printf("(%d° key): ", i);
		for(int j=0; j<LEN; j++){
			index = (i * LEN) + j;
			printf("%02x ", keys[index]);
		}
		printf("\n\n");
	}
}

__host__ void printHeader(long const DK_NUM, long const DK_LEN, int const  BX, int const C){
	printf("\n- - - - - - - - - REQUEST - - - - - - - -\n");
	printf("| Keys:              %15s \t |\n", prettyPrintNumber(DK_NUM));
	printf("| Bytes per Key:     %15s \t |\n", prettyPrintNumber(DK_LEN));
	printf("| Threads per block: %15d \t |\n", BX);
	printf("| H_LEN Bytes:       %15d \t |\n", H_LEN);
	printf("| Iterations:        %15s \t |\n", prettyPrintNumber(C));
	printf("- - - - - - - - - - - - - - - - - - - - -\n\n");
}

__host__ void printKernelDebugInfo(int const K_ID, int const THREAD_X_K, int const K_BYTES_GENERATED, int const DK_LEN){
	printf("%d° kernel, %d thread, generate %d Bytes (%d Bytes each block), derived key len %d\n", K_ID+1, THREAD_X_K, K_BYTES_GENERATED, H_LEN, DK_LEN);
}
